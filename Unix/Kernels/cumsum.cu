#include "hip/hip_runtime.h"
#include "GPUMagic.h"

template <typename T>
__global__ void cumsum_kernel_phase1(matrix<T> *res, matrix<T> *partial, matrix<T> *arr, size_t *chunk_size)
{
    size_t n = arr->get_ncols();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = partial->get_ncols(); 
    size_t len = *chunk_size;

    T sum = 0;
    size_t start = len * tid, end = len * tid + len - 1;
    for(size_t i = start; i <= end; i++){
        sum += arr->at(0, i);
    }
    partial->put(0, tid, sum);
}

template <typename T>
__global__ void cumsum_kernel_phase2(matrix<T> *res, matrix<T> *partial, matrix<T> *arr, size_t *chunk_size)
{
    size_t n = arr->get_ncols();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = partial->get_ncols();
    size_t len = *chunk_size;

    T prev_sum = 0;
    for(int i = 0; i < tid; i++){
        prev_sum += partial->at(0, i);
    }
    size_t start = len * tid, end = len * tid + len - 1;
    T curr_sum = prev_sum;
    for(int i = start; i <= end; i++){
        curr_sum += arr->at(0, i);
        res->put(0, i, curr_sum);
    }
}

template <typename T>
void cumsum (
    matrix<T> **res,
    matrix<T> *arr,
    size_t chunk_size
)
{
    assert(arr->get_storage_type() == DENSE);
    assert(res != NULL);

    int nthreads = ((arr->get_ncols() + chunk_size - 1) / chunk_size);

    size_t *d_chunk_size;
    hipMalloc(&d_chunk_size, sizeof(size_t));
    hipMemcpy(d_chunk_size, &chunk_size, sizeof(size_t), hipMemcpyHostToDevice);

    matrix<T> *h_partial = new matrix<T>(DENSE, CPU);
    h_partial->init(NULL, NULL, NULL, 0, 1, nthreads);

    matrix<T> *h_res = new matrix<T>(DENSE, CPU);
    h_res->init(NULL, NULL, NULL, 0, 1, arr->get_ncols());

    matrix<T> *d_partial = to_gpu(h_partial), *d_res = to_gpu(h_res);
    delete h_partial; delete h_res; 

    matrix<T> *d_arr = to_gpu(arr);

    cumsum_kernel_phase1<<<1, nthreads>>>(d_res, d_partial, d_arr, d_chunk_size);
    CU_TRY(hipPeekAtLastError());
    CU_TRY(hipDeviceSynchronize());

    cumsum_kernel_phase2<<<1, nthreads>>>(d_res, d_partial, d_arr, d_chunk_size);
    CU_TRY(hipPeekAtLastError());
    CU_TRY(hipDeviceSynchronize());

    h_res = to_cpu(d_res);
    (*res) = h_res;

    gpu_del(d_res);
    gpu_del(d_arr);
    gpu_del(d_partial);
    hipFree(d_chunk_size);
}

template void cumsum<float>(matrix<float> **res, matrix<float> *arr, size_t chunk);
template void cumsum<double>(matrix<double> **res, matrix<double> *arr, size_t chunk);
template void cumsum<int>(matrix<int> **res, matrix<int> *arr, size_t chunk);